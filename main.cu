#include "hip/hip_runtime.h"
#include "util.h"

__global__ void cudaFrequencyCounter(int *input, int *counter);

void launchKernel(int *dev_report, int *dev_count) {
	int blocks = TRANSACTIONS_L;
	int threads = ATRIBUTES_C;
	cout << "\nBlocks = " << blocks << " | Threads per Blocks = " << threads << endl;
	cudaFrequencyCounter<<< blocks, threads >>>(dev_report, dev_count);
}

__global__ void cudaFrequencyCounter(int *input, int *counter) {
	int id = 0;
	int tx_col = threadIdx.x;
	int bx_lin = blockIdx.x;
	__shared__ int totalSum[ATRIBUTES_C];
	
	totalSum[tx_col] = 0; counter[tx_col] = 0;
	__syncthreads();
	
	if(tx_col < ATRIBUTES_C && bx_lin < TRANSACTIONS_L) {

		id = (ATRIBUTES_C * bx_lin) + tx_col; 

		if(input[id] == 1)
			atomicAdd(&totalSum[tx_col], 1);

		__syncthreads();
	}

	for(int i = 0; i < ATRIBUTES_C; i++)
		counter[i] = totalSum[i];
}

int main(int argc, char** argv) {
			
	int *dev_count, *dev_report;
	int parallelCounter[ATRIBUTES_C], sequentialCounter[ATRIBUTES_C];
	
	#pragma region Convert threshold to integer 
	if(argc != 3) {
		printf("\nInput: %s minimum_support minimum_confidence\n", argv[0]);
		exit(EXIT_FAILURE); 
	}
	cout << "\nStarting...\n";
	float support = atof(argv[1]), confidence = atof(argv[2]);
	int sup = support * TRANSACTIONS_L, conf = confidence * TRANSACTIONS_L;
	cout << "\nMAIN THRESHOLD " << sup << " " << conf << endl;	
	#pragma endregion 

	#pragma region Allocate memory 
	HANDLE_ERROR(hipMalloc((void**)&dev_count, sizeof(parallelCounter)));
	HANDLE_ERROR(hipMalloc((void**)&dev_report, sizeof(report)));
	HANDLE_ERROR(hipMemcpy(dev_report, report, sizeof(report), hipMemcpyHostToDevice));
	#pragma endregion 

	launchKernel(dev_report, dev_count);

	#pragma region Handle Memory 
	HANDLE_ERROR(hipMemcpy(parallelCounter, dev_count, sizeof(parallelCounter), hipMemcpyDeviceToHost));
	hipFree(dev_report);
	hipFree(dev_count);
	#pragma endregion

	cout << "\n\nFinished.\n\n";
	return 0;
}
